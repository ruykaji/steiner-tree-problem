#include "hip/hip_runtime.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <vector>

#include "mst.hpp"

#define CHECK_CUDA(call)                                                        \
    {                                                                           \
        const hipError_t error = call;                                         \
        if (error != hipSuccess) {                                             \
            printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
            printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
            exit(1);                                                            \
        }                                                                       \
    }

#define CHECK_KERNEL()                                                          \
    {                                                                           \
        const hipError_t error = hipGetLastError();                           \
        if (error != hipSuccess) {                                             \
            printf("Kernel launch failure: %s:%d, ", __FILE__, __LINE__);       \
            printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
            exit(1);                                                            \
        }                                                                       \
    }

struct Edge {
    int32_t destination;
    int32_t weight;
};

void add_edge(std::vector<std::vector<Edge>>& t_graph, int32_t t_a, int32_t t_b, int32_t t_w)
{
    t_graph[t_a - 1].emplace_back(Edge { t_b - 1, t_w });
    t_graph[t_b - 1].emplace_back(Edge { t_a - 1, t_w });
}

// =============================== HELPERS ================================

namespace cuda_array {
template <typename T>
__global__ void set_at(T* t_array, T t_value, size_t t_index) { t_array[t_index] = t_value; }

template <typename T>
__global__ void set(T* t_array, T t_value, size_t t_size)
{
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < t_size; i += stride) {
        t_array[i] = t_value;
    }
}
};

// ========================================================================

__global__ void update_distance(int32_t t_source, int32_t* t_dist, Edge** t_graph)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    t_dist[t_graph[t_source][idx].destination] = t_graph[t_source][idx].weight;
}

__global__ void find_min_distance(size_t t_size, int32_t* t_u, int32_t* t_dist, int32_t* t_processed)
{
    int32_t min_distance = __INT_MAX__;
    *t_u = -1;

    for (int32_t i = 0; i < t_size; ++i) {
        if (t_processed[i] == -1 && t_dist[i] < min_distance) {
            min_distance = t_dist[i];
            *t_u = i;
        }
    }

    if (*t_u != -1) {
        t_processed[*t_u] = 1;
    }
}

__global__ void processed_edges(int32_t t_u, Edge** t_graph, int32_t t_edge_size, int32_t* t_dist, int32_t* t_processed, int32_t t_size)
{
    int32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = index; i < t_edge_size; i += stride) {
        if (t_processed[t_graph[t_u][i].destination] == -1) {
            atomicMin(&t_dist[t_graph[t_u][i].destination], t_dist[t_u] + t_graph[t_u][i].weight);
        };
    }
}

void calculate_distance(int32_t t_size, int32_t t_source, Edge** t_graph, int32_t* t_edges_count, int32_t* t_radius)
{

    int32_t* dist;
    CHECK_CUDA(hipMalloc(&dist, t_size * sizeof(int32_t)));

    int32_t* processed;
    CHECK_CUDA(hipMalloc(&processed, t_size * sizeof(int32_t)));
    cuda_array::set<<<((t_size + 256 - 1) / 256), 256>>>(processed, -1, t_size);
    cuda_array::set_at<<<1, 1>>>(processed, 1, t_source);

    int32_t* host_u = (int32_t*)malloc(sizeof(int32_t));
    int32_t* device_u;
    CHECK_CUDA(hipMalloc(&device_u, sizeof(int32_t)));

    auto start = std::chrono::high_resolution_clock::now();

    cuda_array::set<<<((t_size + 256 - 1) / 256), 256>>>(dist, INT_MAX, t_size);
    cuda_array::set_at<<<1, 1>>>(dist, 0, t_source);

    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "Time taken by function: " << duration.count() << " microseconds" << std::endl;

    int32_t processed_counter = 1;

    update_distance<<<1, t_edges_count[t_source]>>>(t_source, dist, t_graph);

    while (processed_counter < t_size) {
        find_min_distance<<<1, 1>>>(t_size, device_u, dist, processed);
        CHECK_CUDA(hipMemcpy(host_u, device_u, sizeof(int32_t), hipMemcpyDeviceToHost));

        if (*host_u == -1) {
            break;
        }

        ++processed_counter;

        processed_edges<<<((t_edges_count[*host_u] + 256 - 1) / 256), 256>>>(*host_u, t_graph, t_edges_count[*host_u], dist, processed, t_size);
    }


    int32_t* host_dist = (int32_t*)malloc(t_size * sizeof(int32_t));
    hipMemcpy(host_dist, dist, t_size * sizeof(int32_t), hipMemcpyDeviceToHost);

    for (int32_t i = 0; i < t_size; ++i) {
        printf("Value %d: %d\n", i + 1, host_dist[i]);
    }

    // Free cuda
    hipFree(dist);
    hipFree(processed);
    hipFree(device_u);

    // Free host
    free(host_u);
}